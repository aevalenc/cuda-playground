#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2025 Alejandro Valencia
 */

#include "examples/closest_neighbor/closest_neighbor.h"
#include "examples/closest_neighbor/launch.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>
#include <time.h>

double LaunchGPU(std::int32_t number_of_points,
                 std::int32_t N,
                 std::int32_t threads_per_block,
                 std::int32_t blocks_per_grid)
{

    // Allocate device memory for two arrays of double2 points
    double2* h_points = new double2[number_of_points];
    std::int32_t* h_neighbors = new std::int32_t[number_of_points];

    // Initialize the arrays
    for (std::int32_t i = 0; i < number_of_points; ++i)
    {
        h_points[i] = make_double2(static_cast<double>(rand()) / RAND_MAX, static_cast<double>(rand()) / RAND_MAX);
    }
    for (std::int32_t i = 0; i < N; ++i)
    {
        std::cout << "h_point[" << i << "] = (" << h_points[i].x << ", " << h_points[i].y << ")\n";
    }

    double2* device_points;
    std::int32_t* device_neighbors;

    const auto start = clock();
    if (hipMalloc((void**)&device_points, sizeof(double2) * number_of_points) != hipSuccess)
    {
        std::cerr << "Failed to allocate device memory for device_points\n";
        return -1;
    }
    if (hipMalloc((void**)&device_neighbors, sizeof(std::int32_t) * number_of_points) != hipSuccess)
    {
        std::cerr << "Failed to allocate device memory for device_neighbors;\n";
        hipFree(device_points);
        return -1;
    }

    if (hipMemcpy(device_points, h_points, sizeof(double2) * number_of_points, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cerr << "Failed to copy data from host to device for device_points\n";
        hipFree(device_points);
        hipFree(device_neighbors);
        return -1;
    }

    FindClosestNeighborGPU<<<blocks_per_grid, threads_per_block>>>(device_points, device_neighbors, number_of_points);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    if (hipMemcpy(h_neighbors, device_neighbors, sizeof(std::int32_t) * number_of_points, hipMemcpyDeviceToHost) !=
        hipSuccess)
    {
        std::cerr << "Failed to copy data from device to host\n";
        delete[] h_points;
        delete[] h_neighbors;
        hipFree(device_points);
        hipFree(device_neighbors);
        return -1;
    }

    const auto end = clock();
    const double elapsed_time = static_cast<double>(end - start) / CLOCKS_PER_SEC;
    std::cout << "Elapsed GPU time: " << elapsed_time << " seconds\n";

    for (std::int32_t i = 0; i < N; ++i)
    {
        std::cout << "h_neighbors[" << i << "] = " << h_neighbors[i] << "\n";
    }

    delete[] h_points;
    delete[] h_neighbors;
    hipFree(device_points);
    hipFree(device_neighbors);

    hipDeviceReset();

    return elapsed_time;
}

double LaunchCPU(std::int32_t number_of_points, std::int32_t N)
{

    // Allocate device memory for two arrays of double2 points
    double2* h_points = new double2[number_of_points];
    std::int32_t* h_neighbors = new std::int32_t[number_of_points];

    // Initialize the arrays
    for (std::int32_t i = 0; i < number_of_points; ++i)
    {
        h_points[i] = make_double2(static_cast<double>(rand()) / RAND_MAX, static_cast<double>(rand()) / RAND_MAX);
    }

    for (std::int32_t i = 0; i < N; ++i)
    {
        std::cout << "h_point[" << i << "] = (" << h_points[i].x << ", " << h_points[i].y << ")\n";
    }

    const auto start = clock();
    for (std::int32_t idx = 0; idx < number_of_points; ++idx)
    {
        // Initialize the closest neighbor and minimum distance
        double min_distance = std::numeric_limits<double>::max();  // A large initial value

        // Iterate through all points to find the closest neighbor
        for (std::int32_t current_idx = 0; current_idx < number_of_points; ++current_idx)
        {
            if (current_idx != idx)  // Skip self-comparison
            {
                const double distance = pow(h_points[idx].x - h_points[current_idx].x, 2) +
                                        pow(h_points[idx].y - h_points[current_idx].y, 2);
                if (distance < min_distance)
                {
                    min_distance = distance;
                    h_neighbors[idx] = current_idx;  // Store the closest neighbor
                }
            }
        }
    }
    const auto end = clock();
    const double elapsed_time = static_cast<double>(end - start) / CLOCKS_PER_SEC;
    std::cout << "Elapsed CPU time: " << elapsed_time << " seconds\n";

    for (std::int32_t i = 0; i < N; ++i)
    {
        std::cout << "h_neighbor[" << i << "] = " << h_neighbors[i] << "\n";
    }

    delete[] h_points;
    delete[] h_neighbors;

    return elapsed_time;
}
