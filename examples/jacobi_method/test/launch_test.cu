#include "hip/hip_runtime.h"
/*
 * Jacobi Solver with Shared Memory in CUDA
 *
 */

#include "examples/matrix_multiplication/matrix_multiplication.h"
#include "examples/matrix_multiplication/test/launch_test.cuh"
#include "examples/matrix_multiplication/utils.h"
#include <cstdint>
#include <gtest/gtest.h>
#include <time.h>

// void LaunchWithSharedMemory(const int32_t* A,
//                             const int32_t* B,
//                             int32_t* C,
//                             const std::int32_t M,
//                             const std::int32_t N,
//                             const std::int32_t P)
// {  // Allocate device memory
//     std::int32_t* d_A;
//     std::int32_t* d_B;
//     std::int32_t* d_C;

//     const auto start = clock();
//     if (utils::AllocateAndCopyToDevice(d_A, d_B, d_C, A, B, M, N, P) != 0)
//     {
//         return;  // Error already handled in AllocateAndCopyToDevice
//     }

//     // Launch kernel
//     const auto num_blocks_x = (P + kBlockSize - 1) / kBlockSize;
//     const auto num_blocks_y = (M + kBlockSize - 1) / kBlockSize;

//     const auto grid_dim = dim3(num_blocks_x, num_blocks_y, 1);
//     const auto block_dim = dim3(kBlockSize, kBlockSize);

//     printf("Launching kernel with configuration: (%d, %d) x (%d, %d) threads per block\n",
//            num_blocks_x,
//            num_blocks_y,
//            kBlockSize,
//            kBlockSize);

//     MatMultWithSharedMemoryGPU<<<grid_dim, block_dim>>>(d_A, d_B, d_C, M, N, P);

//     CUDA_CHECK(hipGetLastError());
//     CUDA_CHECK(hipDeviceSynchronize());

//     // Copy results back

//     if (hipMemcpy(C, d_C, sizeof(std::int32_t) * M * P, hipMemcpyDeviceToHost) != hipSuccess)
//     {
//         std::cerr << "Failed to copy data from device to host\n";
//         hipFree(d_A);
//         hipFree(d_B);
//         hipFree(d_C);
//         return;
//     }
//     const auto end = clock();
//     const double elapsed_time = static_cast<double>(end - start) / CLOCKS_PER_SEC;
//     std::cout << "Elapsed GPU time: " << elapsed_time << " seconds\n";

//     hipFree(d_A);
//     hipFree(d_B);
//     hipFree(d_C);
// }
