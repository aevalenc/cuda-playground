#include "hip/hip_runtime.h"
/*
 * Jacobi Solver in CUDA
 *
 * Copyright (C) 2025 Name Alejandro Valencia
 */

#include "examples/jacobi_method/1d_grid/jacobi.h"
#include <cstdint>

__global__ void JacobiSolveGPU1D(const double* A, const double* b, double* x0, double* x, std::int32_t N)
{
    // Get Global thread index
    std::int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
    {
        return;  // Out of bounds check
    }

    // Calculate sum
    double sum = 0;
    for (std::int32_t k = 0; k < N; ++k)
    {
        if (k != idx)
        {
            sum += A[k + N * idx] * x0[k];
        }
    }

    // Calculate next iteration
    x[idx] = (b[idx] - sum) / A[idx + N * idx];
}

__global__ void JacobiSolveWithSharedMemoryGPU(const double* A, const double* b, double* x0, double* x, std::int32_t N)
{
    // Calculate the global thread index
    std::int32_t row = blockIdx.y * blockDim.y + threadIdx.y;
    std::int32_t column = blockIdx.x * blockDim.x + threadIdx.x;

    double final_sum = 0.0;
    for (std::int32_t current_block = 0; current_block < (N + kBlockSize - 1) / kBlockSize; ++current_block)
    // for (std::int32_t current_block = 0; current_block < 1; ++current_block)
    {
        // printf("Block (%d,%d) Processing current_block: %d\n", blockIdx.x, blockIdx.y, current_block);
        double inner_sum = 0.0;

        // Shared memory for A and x0
        __shared__ double shared_A[kBlockSize];
        __shared__ double shared_x0[kBlockSize];

        // Load A and B into shared memory
        if (row < N && (current_block * blockDim.x + threadIdx.x < N) && column < kBlockSize)
        {
            // printf("current_block: %d, row: %d, threadIdx.x: %d\n", current_block, row, threadIdx.x);
            shared_A[threadIdx.x] = A[row * N + current_block * blockDim.x + threadIdx.x];
            // printf("Block (%d,%d) Loading A[%d,%d] = %f\n",
            //        blockIdx.x,
            //        blockIdx.y,
            //        row,
            //        current_block * blockDim.x + threadIdx.x,
            //        shared_A[threadIdx.x]);
        }
        else
        {
            shared_A[threadIdx.x] = 0;
        }
        if (column < kBlockSize && current_block * blockDim.y + threadIdx.y < N)
        {
            shared_x0[threadIdx.x] = x0[(current_block * kBlockSize + threadIdx.x)];
            // printf("Block (%d,%d) Loading x0[%d] = %f\n",
            //        blockIdx.x,
            //        blockIdx.y,
            //        (current_block * kBlockSize + threadIdx.x),
            //        shared_x0[threadIdx.x]);
        }
        else
        {
            shared_x0[threadIdx.x] = 0;
        }

        __syncthreads();

        // Perform the multiplication
        if (row < N && column < 1)
        {
            // double sum = 0.0;
            for (std::int32_t j = 0; j < kBlockSize; ++j)
            {
                std::int32_t current_index = current_block * kBlockSize + j;
                // printf("Block (%d,%d) Current index: %d, row: %d, column: %d\n",
                //        blockIdx.x,
                //        blockIdx.y,
                //        current_index,
                //        row,
                //        column);
                if (row != current_index && current_index < N)  // Avoid self-multiplication
                {
                    inner_sum += shared_A[j] * shared_x0[j];
                    // printf("Block (%d,%d) Multiplying A[%d,%d] * x0[%d] = %f * %f\n",
                    //        blockIdx.x,
                    //        blockIdx.y,
                    //        row,
                    //        current_block * blockDim.x + threadIdx.x,
                    //        current_index,
                    //        shared_A[j],
                    //        shared_x0[j]);
                }
            }
            final_sum += inner_sum;
        }

        __syncthreads();
    }

    // Calculate next iteration
    if (row < N && column < 1)
    {
        x[row] = (b[row] - final_sum) / A[row + N * row];
    }
}
