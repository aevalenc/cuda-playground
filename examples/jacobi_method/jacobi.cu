#include "hip/hip_runtime.h"
/*
 * add_numbers.cu
 *
 * Copyright (C) 2025 Name Alejandro Valencia
 */

#include "examples/jacobi_method/jacobi.h"
#include <cstdint>

__global__ void JacobiSolveGPU(double* A, double* b, double* xn, double* x, std::int32_t N)
{
    // Get Global thread index
    std::int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate sum
    double sum = 0;
    for (std::int32_t k = 0; k < N; ++k)
    {
        if (k != idx)
        {
            sum += A[k + N * idx] * xn[k];
        }
    }

    // Calculate next iteration
    x[idx] = (b[idx] - sum) / A[idx + N * idx];
}
