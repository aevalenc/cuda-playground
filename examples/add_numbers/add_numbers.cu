/*
 * add_numbers.cu
 *
 * Copyright (C) 2025 Name Alejandro Valencia
 */

#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(expr)                                                                                     \
    do                                                                                                       \
    {                                                                                                        \
        hipError_t err = (expr);                                                                            \
        if (err != hipSuccess)                                                                              \
        {                                                                                                    \
            fprintf(stderr, "CUDA Error Code  : %d\n     Error String: %s\n", err, hipGetErrorString(err)); \
            exit(err);                                                                                       \
        }                                                                                                    \
    } while (0)

__global__ void AddIntsCUDA(int* a, int* b)
{
    a[0] += b[0];
}

void Launch()
{
    int a = 5;
    int b = 10;
    int* d_a;
    int* d_b;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    AddIntsCUDA<<<1, 1>>>(d_a, d_b);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "The answer is " << a << "\n";

    hipFree(d_a);
    hipFree(d_b);
}
