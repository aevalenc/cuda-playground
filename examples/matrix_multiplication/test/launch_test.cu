/*
 * Closest Neighbor Search using Shared Memory in CUDA Test
 */

#include "examples/matrix_multiplication/matrix_multiplication.h"
#include "examples/matrix_multiplication/test/launch_test.cuh"
#include "examples/matrix_multiplication/utils.h"
#include <cstdint>
#include <gtest/gtest.h>

void Launch(const int32_t* A,
            const int32_t* B,
            int32_t* C,
            const std::int32_t M,
            const std::int32_t N,
            const std::int32_t P)
{

    // Allocate device memory
    std::int32_t* d_A;
    std::int32_t* d_B;
    std::int32_t* d_C;
    hipMalloc(&d_A, sizeof(std::int32_t) * M * N);
    hipMalloc(&d_B, sizeof(std::int32_t) * N * P);
    hipMalloc(&d_C, sizeof(std::int32_t) * M * P);

    // Copy points to device
    hipMemcpy(d_A, A, sizeof(std::int32_t) * M * N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(std::int32_t) * N * P, hipMemcpyHostToDevice);

    // Launch kernel (4 threads per block)
    const auto num_blocks = (M + kTestBlockSize - 1) / kTestBlockSize;
    printf("Launching kernel with %d blocks of %d threads each\n", num_blocks, kTestBlockSize);
    MatMultGPU<<<num_blocks, kTestBlockSize>>>(d_A, d_B, d_C, M, N, P);
    hipDeviceSynchronize();

    // Copy results back
    hipMemcpy(C, d_C, sizeof(std::int32_t) * M * P, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void LaunchAccelerated(const int32_t* A,
                       const int32_t* B,
                       int32_t* C,
                       const std::int32_t M,
                       const std::int32_t N,
                       const std::int32_t P)
{

    // Allocate device memory
    std::int32_t* d_A;
    std::int32_t* d_B;
    std::int32_t* d_C;
    hipMalloc(&d_A, sizeof(std::int32_t) * M * N);
    hipMalloc(&d_B, sizeof(std::int32_t) * N * P);
    hipMalloc(&d_C, sizeof(std::int32_t) * M * P);

    // Copy points to device
    if (hipMemcpy(d_A, A, sizeof(std::int32_t) * M * N, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cerr << "Failed to copy data from host to device for Matrix A\n";
        hipFree(d_A);
        hipFree(d_B);
        return;
    }
    if (hipMemcpy(d_B, B, sizeof(std::int32_t) * N * P, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cerr << "Failed to copy data from host to device for Matrix A\n";
        hipFree(d_A);
        hipFree(d_B);
        return;
    }

    // Launch kernel
    const auto num_blocks_x = (P + kTestBlockSize - 1) / kTestBlockSize;
    const auto num_blocks_y = (M + kTestBlockSize - 1) / kTestBlockSize;

    printf("Launching kernel with %d blocks in x, %d blocks in y, and %d threads per block\n",
           num_blocks_x,
           num_blocks_y,
           kTestBlockSize);

    AccelMatMultGPU<<<dim3(num_blocks_x, num_blocks_y), dim3(kTestBlockSize, kTestBlockSize)>>>(d_A, d_B, d_C, M, N, P);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy results back

    if (hipMemcpy(C, d_C, sizeof(std::int32_t) * M * P, hipMemcpyDeviceToHost) != hipSuccess)
    {
        std::cerr << "Failed to copy data from device to host\n";
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
